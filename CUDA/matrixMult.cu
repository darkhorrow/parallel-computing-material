#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>

__global__ void matrixMult(double *a, double *b, double *c, int n) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    if (row < n && col < n) {
        for (int i = 0; i < n; ++i) {
            c[row*n + col] += a[row*n + i] * b[i*n + col];
        }
    }
}

int main(int argc, char* argv) {
    int N = 320;
    int size = N*N; // Matrix size
	int blocksPerGrid = 32; // Blocks per CUDA grid

    double *a, *b, *c;          // Host copies of matrix a, b, c
    double *d_a, *d_b, *d_c;    // Device copies of a, b, c

    // Alloc memory in host for double array
    a = (double*) malloc(size*sizeof(double));
    b = (double*) malloc(size*sizeof(double));
    c = (double*) malloc(size*sizeof(double));
    
    // Init a simple matrix in vector format
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            a[i*N + j] = i*N + j;
            b[i*N + j] = i*N + j;
        }
    }

    // Alloc memory in device for double array
    hipMalloc(&d_a, size*sizeof(double)); 
    hipMalloc(&d_b, size*sizeof(double));
    hipMalloc(&d_c, size*sizeof(double));

    // Copy from host to device
    hipMemcpy(d_a, a, size*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*sizeof(double),hipMemcpyHostToDevice);

    
    // Nº of threads and blocks to use
    dim3 blocksPerGrid( blocksPerGrid, blocksPerGrid);
    dim3 threadsPerBlock( N / blocksPerGrid, N / blocksPerGrid);

    hipEvent_t start, stop;
    float elapsedTime;

	// Start recording time elapsed
    hipEventCreate(&start);
    hipEventRecord(start,0);

	// Launch kernel
    matrixMult<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_c,N);
	
	// Wait until the kernel has finished its execution
    hipDeviceSynchronize();

	// Stop recording time elapsed
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time : %f ms\n" ,elapsedTime);

    // Copy from device to host the result
    hipMemcpy(c, d_c, size*sizeof(double), hipMemcpyDeviceToHost);


	// Uncomment this block to visualize the result
	
	/*
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            printf("%f \t", c[i*N + j]);
        }
        printf("\n");
    }
	*/

    // Free memory used in both host and device
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    printf("Done\n");
    return 0;
}

