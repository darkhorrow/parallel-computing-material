#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>


__global__ void mult(double *a, double *b, double *result, int n) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < n && col < n) {
        for (int i = 0; i < n; ++i) {
            result[row * n + col] += a[row * n + i] * b[i * n + col];
        }
    }

}



int main(void) {

    srand (1);
    int dimension = 320;
    int size = dimension*dimension;

    double *a, *b, *result;          // host copies of matrix a,b,result
    double *d_a, *d_b, *d_result;    // device copies of a,b,result

    // memoria host
    a = (double *)malloc(size*sizeof(double));
    b = (double *)malloc(size*sizeof(double));
    result = (double *)malloc(size*sizeof(double));
    
    // inicializamos matrices
    for(int i = 0; i < dimension; i++) {
        for(int j = 0; j < dimension; j++) {
            a[i*dimension+j] = i*dimension+j;
            b[i*dimension+j] = i*dimension+j;
            
        }
    }

    // memoria device
    hipMalloc(&d_a, size*sizeof(double)); 
    hipMalloc(&d_b, size*sizeof(double));
    hipMalloc(&d_result, size*sizeof(double));

    // copiamos a device
    hipMemcpy(d_a,a,size*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size*sizeof(double),hipMemcpyHostToDevice);

    int blockSize = 32;
    // llamamos al kernel
    dim3 blocksPerGrid( blockSize, blockSize);
    dim3 threadsPerBlock( dimension / blockSize, dimension / blockSize);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    mult<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_result,dimension);
    hipDeviceSynchronize();

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time : %f ms\n" ,elapsedTime);

    // ???????????
    hipMemcpy(result, d_result, size*sizeof(double), hipMemcpyDeviceToHost);


    // for(int i = 0; i < dimension; i++) {
    //     for(int j = 0; j < dimension; j++) {
    //         printf("%f \t", result[i*dimension+j]);
    //     }
    //     printf("\n");
    // }

    // ????????
    free(a); free(b); free(result);
    hipFree(d_a); hipFree(d_b); hipFree(d_result);

    printf("Done\n");
    return 0;
}

